#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 * Full license terms provided in LICENSE.md file.
 */

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <NvInfer.h>
#include <opencv2/opencv.hpp>
#include "examples/classify_image/utils.h"


using namespace std;
using namespace nvinfer1;


class Logger : public ILogger
{
  void log(Severity severity, const char * msg) override
  {
    if (severity != Severity::kINFO)
      cout << msg << endl;
  }
} gLogger;


/**
 * image_file: path to image
 * plan_file: path of the serialized engine file
 * label_file: file with <class_name> per line
 * input_name: name of the input tensor
 * output_name: name of the output tensor
 * preprocessing_fn: 'vgg' or 'inception'
 */
int main(int argc, char *argv[])
{
  if (argc != 7)
  {
    cout << "Usage: classify_image <image_file> <plan_file> <label_file> <input_name> <output_name> <preprocessing_fn>\n";
    return 0;
  }

  string imageFilename = argv[1];
  string planFilename = argv[2];
  string labelFilename = argv[3];
  string inputName = argv[4];
  string outputName = argv[5];
  string preprocessingFn = argv[6];

  /* load the engine */
  cout << "Loading TensorRT engine from plan file..." << endl;
  ifstream planFile(planFilename); 

  if (!planFile.is_open())
  {
    cout << "Could not open plan file." << endl;
    return 1;
  }

  stringstream planBuffer;
  planBuffer << planFile.rdbuf();
  string plan = planBuffer.str();
  IRuntime *runtime = createInferRuntime(gLogger);
  ICudaEngine *engine = runtime->deserializeCudaEngine((void*)plan.data(), plan.size(), nullptr);
  IExecutionContext *context = engine->createExecutionContext();
  
  /* get the input / output dimensions */
  int inputBindingIndex, outputBindingIndex;
  inputBindingIndex = engine->getBindingIndex(inputName.c_str());
  outputBindingIndex = engine->getBindingIndex(outputName.c_str());

  if (inputBindingIndex < 0)
  {
    cout << "Invalid input name." << endl;
    return 1;
  }

  if (outputBindingIndex < 0)
  {
    cout << "Invalid output name." << endl;
    return 1;
  }

  Dims inputDims, outputDims;
  inputDims = engine->getBindingDimensions(inputBindingIndex);
  outputDims = engine->getBindingDimensions(outputBindingIndex);
  int inputWidth, inputHeight;
  inputHeight = inputDims.d[1];
  inputWidth = inputDims.d[2];

  /* read image, convert color, and resize */
  cout << "Preprocessing input..." << endl;
  cv::Mat image = cv::imread(imageFilename, CV_LOAD_IMAGE_COLOR);

  if (image.data == NULL)
  {
    cout << "Could not read image from file." << endl;
    return 1;
  }

  cv::cvtColor(image, image, cv::COLOR_BGR2RGB, 3);
  cv::resize(image, image, cv::Size(inputWidth, inputHeight));

  /* convert from uint8+NHWC to float+NCHW */
  float *inputDataHost, *outputDataHost;
  size_t numInput, numOutput;
  numInput = numTensorElements(inputDims);
  numOutput = numTensorElements(outputDims);
  inputDataHost = (float*) malloc(numInput * sizeof(float));
  outputDataHost = (float*) malloc(numOutput * sizeof(float));
  cvImageToTensor(image, inputDataHost, inputDims);
  if (preprocessingFn == "vgg")
    preprocessVgg(inputDataHost, inputDims);
  else if (preprocessingFn == "inception")
    preprocessInception(inputDataHost, inputDims);
  else
  {
    cout << "Invalid preprocessing function argument, must be vgg or inception. \n" << endl;
    return 1;
  }

  /* transfer to device */
  float *inputDataDevice, *outputDataDevice;
  hipMalloc(&inputDataDevice, numInput * sizeof(float));
  hipMalloc(&outputDataDevice, numOutput * sizeof(float));
  hipMemcpy(inputDataDevice, inputDataHost, numInput * sizeof(float), hipMemcpyHostToDevice);
  void *bindings[2];
  bindings[inputBindingIndex] = (void*) inputDataDevice;
  bindings[outputBindingIndex] = (void*) outputDataDevice;

  /* execute engine */
  cout << "Executing inference engine..." << endl;
  const int kBatchSize = 1;
  context->execute(kBatchSize, bindings);

  /* transfer output back to host */
  hipMemcpy(outputDataHost, outputDataDevice, numOutput * sizeof(float), hipMemcpyDeviceToHost);

  /* parse output */
  vector<size_t> sortedIndices = argsort(outputDataHost, outputDims);

  cout << "\nThe top-5 indices are: ";
  for (int i = 0; i < 5; i++)
    cout << sortedIndices[i] << " ";

  ifstream labelsFile(labelFilename);

  if (!labelsFile.is_open())
  {
    cout << "\nCould not open label file." << endl;
    return 1;
  }

  vector<string> labelMap;
  string label;
  while(getline(labelsFile, label))
  {
    labelMap.push_back(label);
  }

  cout << "\nWhich corresponds to class labels: ";
  for (int i = 0; i < 5; i++)
    cout << endl << i << ". " << labelMap[sortedIndices[i]];
  cout << endl;

  /* clean up */
  runtime->destroy();
  engine->destroy();
  context->destroy();
  free(inputDataHost);
  free(outputDataHost);
  hipFree(inputDataDevice);
  hipFree(outputDataDevice);

  return 0;
}
